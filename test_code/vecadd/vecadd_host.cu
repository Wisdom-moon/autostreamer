#include <omp.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void my_kernel
 ( int inputLength, float * hostOutput, float * hostInput1, float * hostInput2, int length)
{

int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i >= length)
  return;

  {
    hostOutput[i] = hostInput1[i] + hostInput2[i];
  }
}

int main(int argc, char *argv[]) {
  int nstreams = 2;
  hipSetDevice(0);
  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  hipStream_t *streams = (hipStream_t*) malloc(nstreams*sizeof(hipStream_t));
  for (int i = 0; i < nstreams; i++) {
    hipStreamCreate(&(streams[i]));
  }

  hipEvent_t start_event, stop_event;
  int eventflags = hipEventBlockingSync;
  hipEventCreateWithFlags(&start_event, eventflags);
  hipEventCreateWithFlags(&stop_event, eventflags);
  int inputLength = 32;
  int inputLengthBytes;
  inputLengthBytes = inputLength * sizeof(float);
  float *hostInput1;
  float *hostInput2;
  float * d_hostInput1;
  hipMalloc((void **)&d_hostInput1, (inputLengthBytes));
  float * d_hostInput2;
  hipMalloc((void **)&d_hostInput2, (inputLengthBytes));
  float * d_hostOutput;
  hipMalloc((void **)&d_hostOutput, (inputLengthBytes));
  float *hostOutput;

  if (argc > 1) {
    inputLength = atoi (argv[1]);
  }

  hostInput1 = (float *)malloc(inputLengthBytes);
  hostInput2 = (float *)malloc(inputLengthBytes);
  hostOutput       = (float *)malloc(inputLengthBytes);

  for(int i=0; i<inputLength; i++)
  {
    hostInput1[i] = (float)(i % 11);
    hostInput2[i] = (float)(i % 31);
  }


int threadsPerBlock = 8;
for (int i = 0; i < nstreams; i++)
{
int blocksPerGrid = ((((inputLength)-1)-0 + 1)+threadsPerBlock - 1)/(nstreams*threadsPerBlock);
  hipMemcpyAsync(d_hostInput1+i*(((inputLength)-1)-0 + 1)/nstreams, hostInput1+i*(((inputLength)-1)-0 + 1)/nstreams, (inputLengthBytes)/nstreams, hipMemcpyHostToDevice, streams[i]);
  hipMemcpyAsync(d_hostInput2+i*(((inputLength)-1)-0 + 1)/nstreams, hostInput2+i*(((inputLength)-1)-0 + 1)/nstreams, (inputLengthBytes)/nstreams, hipMemcpyHostToDevice, streams[i]);
  my_kernel<<<blocksPerGrid, threadsPerBlock,0, streams[i]>>>(inputLength, d_hostOutput+i*(((inputLength)-1)-0 + 1)/nstreams, d_hostInput1+i*(((inputLength)-1)-0 + 1)/nstreams, d_hostInput2+i*(((inputLength)-1)-0 + 1)/nstreams, (((inputLength)-1)-0 + 1)/nstreams);
  hipMemcpyAsync(hostOutput+i*(((inputLength)-1)-0 + 1)/nstreams, d_hostOutput+i*(((inputLength)-1)-0 + 1)/nstreams, (inputLengthBytes)/nstreams, hipMemcpyDeviceToHost, streams[i]);
}
hipEventRecord(stop_event, 0);
hipEventSynchronize(stop_event);

  for (int i = 0; i < nstreams; i++) {
    hipStreamDestroy(streams[i]);
  }
  hipEventDestroy(start_event);
  hipEventDestroy(stop_event);
  hipFree(d_hostInput1);
  hipFree(d_hostInput2);
  hipFree(d_hostOutput);
  hipDeviceReset();
  for(int i=0; i<inputLength; i++)
  {
    printf("%f  ", hostOutput[i]);
  }
  printf("\n");

  // release host memory
  if(hostInput1 != NULL) free(hostInput1);
  if(hostInput2 != NULL) free(hostInput2);
  if(hostOutput != NULL) free(hostOutput);

  return 0;
}

